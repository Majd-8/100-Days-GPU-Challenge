#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixAdditionKernel(float *matrixA, float *matrixB, float *matrixC, int N) 
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < N) {
        matrixC[row * N + col] = matrixA[row * N + col] + matrixB[row * N + col];
    }
}

int main() {
    const int N = 10;

    float *A_h = new float[N*N];
    float *B_h = new float[N*N];
    float *C_h = new float[N*N];

    for(int i  = 0; i < N; ++i)
    {
        for(int j  = 0; j < N; ++j)
        {
            A_h[i * N + j] = i * N + j;
            B_h[i * N + j] = 10;
        }
    }   

    float *A_d, *B_d, *C_d;
    int size = N*N * sizeof(float);

    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    hipMemcpy(C_d, C_h, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(8, 4, 1);
    dim3 dimGrid(ceil(N / 8.0f), ceil(N / 4.0f), 1);

    matrixAdditionKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    printf("A =\n");
    for(int i = 0; i < N; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            printf("%f\t", A_h[i * N + j]);
        }
        printf("\n");
    }

    printf("B =\n");
    for(int i = 0; i < N; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            printf("%f\t", B_h[i * N + j]);
        }
        printf("\n");
    }

    printf("C =\n");
    for(int i = 0; i < N; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            printf("%f\t", C_h[i * N + j]);
        }
        printf("\n");
    }

    return 0;
}